#include "hip/hip_runtime.h"
// Given a list (lst) of length n
// Output its sum = lst[0] + lst[1] + ... + lst[n-1];

#include <wb.h>

#define BLOCK_SIZE 1024

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

// Issue: within one block, the sum is not being computed correctly!
__global__ void total(float *input, float *output, int len) {
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * BLOCK_SIZE * 2 + tid; // changed

  __shared__ float sdata[BLOCK_SIZE];

  float val = i < len ? input[i] : 0; // If index is within scope of input, load it to smem
  val = i + BLOCK_SIZE < len ? val + input[i + BLOCK_SIZE] : val; // added
  sdata[tid] = val;

  __syncthreads();

  for (unsigned int s = blockDim.x/2; s > 0; s /= 2) {

    if (tid < s) {
      sdata[tid] += sdata[tid + s];
    }

    __syncthreads();
  }
  if (tid == 0) {
    output[blockIdx.x] = sdata[0];
  }
}

int main(int argc, char **argv) {
  int ii;
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  float *deviceInput;
  float *deviceOutput;
  int numInputElements;  // number of elements in the input list
  int numOutputElements; // number of elements in the output list

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput =
      (float *)wbImport(wbArg_getInputFile(args, 0), &numInputElements);

  numOutputElements = numInputElements / (BLOCK_SIZE << 1);
  if (numInputElements % (BLOCK_SIZE << 1)) {
    numOutputElements++;
  }
  printf("Number of output elements: %d \n", numOutputElements);
  hostOutput = (float *)malloc(numOutputElements * sizeof(float));

  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The number of input elements in the input is ",
        numInputElements);
  wbLog(TRACE, "The number of output elements in the input is ",
        numOutputElements);

  wbTime_start(GPU, "Allocating GPU memory.");
  hipMalloc((void **) &deviceInput, numInputElements * sizeof(float));
  hipMalloc((void **) &deviceOutput, numOutputElements * sizeof(float));

  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  hipMemcpy(deviceInput, hostInput, numInputElements * sizeof(float), hipMemcpyHostToDevice);

  wbTime_stop(GPU, "Copying input memory to the GPU.");
  dim3 DimGrid(numOutputElements, 1, 1);
  dim3 DimBlock(BLOCK_SIZE, 1, 1);

  wbTime_start(Compute, "Performing CUDA computation");
  total<<<DimGrid, DimBlock>>>(deviceInput, deviceOutput, numInputElements);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  hipMemcpy(hostOutput, deviceOutput, numOutputElements * sizeof(float), hipMemcpyDeviceToHost);

  wbTime_stop(Copy, "Copying output memory to the CPU");

  /********************************************************************
   * Reduce output vector on the host
   * NOTE: One could also perform the reduction of the output vector
   * recursively and support any size input. 
   ********************************************************************/
   printf("beginning hostOutput value %f \n", hostOutput[0]);
  for (ii = 1; ii < numOutputElements; ii++) {
    printf("hostOutput value %f \n", hostOutput[ii]);
    hostOutput[0] += hostOutput[ii];
  }

  wbTime_start(GPU, "Freeing GPU Memory");
  hipFree(deviceInput);
  hipFree(deviceOutput);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, 1);

  free(hostInput);
  free(hostOutput);

  return 0;
}
